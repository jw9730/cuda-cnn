#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define DEBUG
#define THREADS_PER_BLOCK 512
#define INDEX_ROW_MAJOR_2(i, j, I, J) ((j) + (J) * (i))
#define INDEX_ROW_MAJOR_3(i, j, k, I, J, K) ((k) + (K) * ((j) + (J) * (i)))
#define INDEX_ROW_MAJOR_4(i, j, k, l, I, J, K, L) ((l) + (L) * ((k) + (K) * ((j) + (J) * (i))))

#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))
static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString( err ), file, line);
        exit(EXIT_FAILURE);
    }
}





__global__ void conv_ws(float *I, float *K, float *R, int iw, int ih, int ow, int oh, int kw, int kh, int sw, int sh, int ic, int oc){
    // weight stationary
    int BLOCKS_PER_CHANNEL = ceil(float(ow * oh)/float(THREADS_PER_BLOCK));
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int pid = bid % BLOCKS_PER_CHANNEL; // pixel block index (within channel)
    int cid = bid / BLOCKS_PER_CHANNEL; // output channel index
    // declare on-chip shared memory
    extern __shared__ float M[];
    // read input data once per block (shared across threads)
    // this process could serve as bottleneck, load distribution is critical
    // distribute indices across threads
    int f = kw*kh*ic;
    int load_per_thread = ceil(float(f)/float(THREADS_PER_BLOCK));
    int l = load_per_thread * tid;
    int u = load_per_thread * (tid + 1);
    if (l < f) {
        for (int idx=l; idx<((u<f)?u:f); idx++){
            int i = idx/ic/kh;
            int j = idx/ic%kh;
            int k = idx%ic;
            M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] = K[INDEX_ROW_MAJOR_4(i,j,k,cid, kw,kh,ic,oc)];
        }
    }
    // wait until data is ready
    __syncthreads();
    // compute block index in output pixel dimension
    int ofs = pid * THREADS_PER_BLOCK;
    // handle boundary
    if (tid >= ((ow * oh - ofs < THREADS_PER_BLOCK)? (ow * oh - ofs) : THREADS_PER_BLOCK)) return;
    // retrieve output pixel
    int w = (ofs+tid)/oh;
    int h = (ofs+tid)%oh;
    int w_ofs = w*sw;
    int h_ofs = h*sh;
    float acc = 0;
    // apply convolution
    for (int i=0; i<kw; i++){
        for (int j=0; j<kh; j++){
            for (int k=0; k<ic; k++){
                acc += I[INDEX_ROW_MAJOR_3(w_ofs+i,h_ofs+j,k, iw,ih,ic)] * M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)];
            }
        }
    }
    R[INDEX_ROW_MAJOR_3(w,h,cid, ow,oh,oc)] = acc;
}
__global__ void conv_is(float *I, float *K, float *R, int iw, int ih, int ow, int oh, int kw, int kh, int sw, int sh, int ic, int oc){
    // input stationary
    int BLOCKS_PER_PIXEL = ceil(float(oc)/float(THREADS_PER_BLOCK));
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int cid = bid % BLOCKS_PER_PIXEL; // channel block index (within pixel)
    int pid = bid / BLOCKS_PER_PIXEL; // pixel index
    // compute output pixel of the block
    int h = pid % oh;
    int w = pid / oh;
    int w_ofs = w*sw;
    int h_ofs = h*sh;
    // declare on-chip shared memory
    extern __shared__ float M[];
    // read input data once per block (shared across threads)
    // this process could serve as bottleneck, load distribution is critical
    // distribute indices across threads
    int f = kw*kh*ic;
    int load_per_thread = ceil(float(f)/float(THREADS_PER_BLOCK));
    int l = load_per_thread * tid;
    int u = load_per_thread * (tid + 1);
    if (l < f) {
        for (int idx=l; idx<((u<f)?u:f); idx++){
            int i = idx/ic/kh;
            int j = idx/ic%kh;
            int k = idx%ic;
            M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] = I[INDEX_ROW_MAJOR_3(w_ofs+i,h_ofs+j,k, iw,ih,ic)];
        }
    }
    // wait until data is ready
    __syncthreads();
    // compute block index in output channel dimension
    int ofs = cid * THREADS_PER_BLOCK;
    // handle boundary
    if (tid >= ((oc - ofs < THREADS_PER_BLOCK)? (oc - ofs) : THREADS_PER_BLOCK)) return;
    // apply convolution
    float acc = 0;
    for (int i=0; i<kw; i++){
        for (int j=0; j<kh; j++){
            for (int k=0; k<ic; k++){
                acc += M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] * K[INDEX_ROW_MAJOR_4(i,j,k,ofs+tid, kw,kh,ic,oc)];
            }
        }
    }
    R[INDEX_ROW_MAJOR_3(w,h,ofs+tid, ow,oh,oc)] = acc;
}
extern "C"
void conv2d(float * I, float * K, float * R, int iw, int ih, int ow, int oh, int kw, int kh, int sw, int sh, int ic, int oc) {
    float *dev_I, *dev_K, *dev_R;
    // I: (iw * ih * ic), row major ordered
    // K: (kw * kh * ic * oc), row major ordered
    // R: (ow * oh * oc), row major ordered
    // todo: 2d convolution between I and K
    // loop over outer dimensions, and compute dot product in chunks of size 512
    // kernel function: convolution for a single sliding window
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_I, iw * ih * ic * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_K, kw * kh * ic * oc * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_R, ow * oh * oc * sizeof(float) ) );
    // copy the arrays to the GPU
    HANDLE_ERROR( hipMemcpy( dev_I, I, iw * ih * ic * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_K, K, kw * kh * ic * oc * sizeof(float), hipMemcpyHostToDevice ) );
    // how to organize blocks?
    // maximizing data reuse and parallelism within a block
    // dynamic on-chip memory allocation
    int BLOCK_MEMSIZE = kw * kh * ic * sizeof(float);
    if (ow*oh > 100 * THREADS_PER_BLOCK){
        // weight stationary
        // within a block, hold kernel and thread over output pixels
        int BLOCKS_PER_CHANNEL = ceil(float(ow*oh)/float(THREADS_PER_BLOCK));
        conv_ws<<<oc*BLOCKS_PER_CHANNEL,THREADS_PER_BLOCK,BLOCK_MEMSIZE>>>(dev_I, dev_K, dev_R, iw, ih, ow, oh, kw, kh, sw, sh, ic, oc);
    }else{
        // input stationary
        // within a block, hold input and thread over output channels
        int BLOCKS_PER_PIXEL = ceil(float(oc)/float(THREADS_PER_BLOCK));
        conv_is<<<ow*oh*BLOCKS_PER_PIXEL,THREADS_PER_BLOCK,BLOCK_MEMSIZE>>>(dev_I, dev_K, dev_R, iw, ih, ow, oh, kw, kh, sw, sh, ic, oc);
    }
    // copy the array back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( R, dev_R, ow * oh * oc * sizeof(float), hipMemcpyDeviceToHost ) );
    // cleanup
    hipFree(dev_I); hipFree(dev_K); hipFree(dev_R);
}






__global__ void badd(float *I, float *B, float *R, int ow, int oh, int oc){
    int BLOCKS_PER_CHANNEL = ceil(float(ow * oh)/float(THREADS_PER_BLOCK));
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int pid = bid % BLOCKS_PER_CHANNEL; // pixel block index (within channel)
    int cid = bid / BLOCKS_PER_CHANNEL; // channel index
    // compute block index in output pixel dimension
    int ofs = pid * THREADS_PER_BLOCK;
    // handle boundary
    if (tid >= ((ow * oh - ofs < THREADS_PER_BLOCK)? (ow * oh - ofs) : THREADS_PER_BLOCK)) return;
    // import channelwise parameters to shared memory
    __shared__ float Mem[1];
    if(tid == 0) Mem[0] = B[cid];
    // wait until data is ready
    __syncthreads();
    // add
    ofs = INDEX_ROW_MAJOR_3((ofs + tid)/oh,(ofs + tid)%oh,cid, ow,oh,oc);
    R[ofs] = I[ofs] + Mem[0];
}
extern "C"
void bias_add(float * I, float * B, float * R, int ow, int oh, int oc) {
    float *dev_I, *dev_B, *dev_R;
    // I: (ow * oh * oc), row major ordered
    // B: (oc)
    // R: (ow * oh * oc), row major ordered
    // todo: element-wise addition
    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_I, ow * oh * oc * sizeof(float) );
    hipMalloc( (void**)&dev_B, oc * sizeof(float) );
    hipMalloc( (void**)&dev_R, ow * oh * oc * sizeof(float) );
    // copy the arrays to the GPU
    hipMemcpy( dev_I, I, ow * oh * oc * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_B, B, oc * sizeof(float), hipMemcpyHostToDevice );
    // block = channel, thread over pixels
    int BLOCKS_PER_CHANNEL = ceil(float(ow*oh)/float(THREADS_PER_BLOCK));
    badd<<<oc*BLOCKS_PER_CHANNEL,THREADS_PER_BLOCK>>>(dev_I, dev_B, dev_R, ow, oh, oc);
    // copy the array back from the GPU to the CPU
    hipMemcpy( R, dev_R, ow * oh * oc * sizeof(float), hipMemcpyDeviceToHost );
    // cleanup
    hipFree(dev_I); hipFree(dev_B); hipFree(dev_R);
}






__global__ void lr(float *I, float *R, int ow, int oh, int oc){
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    // handle boundary
    int ofs = ow*oh*oc - bid*THREADS_PER_BLOCK;
    if (tid >= (ofs < THREADS_PER_BLOCK? ofs : THREADS_PER_BLOCK)) return;
    // add
    ofs = bid*THREADS_PER_BLOCK+tid;
    float input = I[ofs];
    R[ofs] = (input > 0)? input : input * 0.1f;
}
extern "C"
void leaky_relu(float * I, float * R, int ow, int oh, int oc) {
    float *dev_I, *dev_R;
    // I: (ow * oh * oc), row major ordered
    // R: (ow * oh * oc), row major ordered
    // todo: element-wise rectification
    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_I, ow * oh * oc * sizeof(float) );
    hipMalloc( (void**)&dev_R, ow * oh * oc * sizeof(float) );
    // copy the arrays to the GPU
    hipMemcpy( dev_I, I, ow * oh * oc * sizeof(float), hipMemcpyHostToDevice );
    // block = channel, thread over pixels
    int BLOCKS = ceil(float(ow*oh*oc)/float(THREADS_PER_BLOCK));
    lr<<<BLOCKS,THREADS_PER_BLOCK>>>(dev_I, dev_R, ow, oh, oc);
    // copy the array back from the GPU to the CPU
    hipMemcpy( R, dev_R, ow * oh * oc * sizeof(float), hipMemcpyDeviceToHost );
    // cleanup
    hipFree(dev_I); hipFree(dev_R);
}





__global__ void bn(float *I, float *M, float *G, float *V, float *R, float eps, int ow, int oh, int oc){
    int BLOCKS_PER_CHANNEL = ceil(float(ow * oh)/float(THREADS_PER_BLOCK));
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int pid = bid % BLOCKS_PER_CHANNEL; // pixel block index (within channel)
    int cid = bid / BLOCKS_PER_CHANNEL; // channel index
    // compute block index in output pixel dimension
    int ofs = pid * THREADS_PER_BLOCK;
    // handle boundary
    if (tid >= ((ow * oh - ofs < THREADS_PER_BLOCK)? (ow * oh - ofs) : THREADS_PER_BLOCK)) return;
    // import channelwise parameters to shared memory
    __shared__ float memory[3];
    if(tid == 0){
        memory[0] = G[cid];
        memory[1] = M[cid];
        memory[2] = V[cid];
    }
    // wait until data is ready
    __syncthreads();
    // retrieve output pixel
    ofs = INDEX_ROW_MAJOR_3((ofs + tid)/oh,(ofs + tid)%oh,cid, ow,oh,oc);
    // normalize
    R[ofs] = memory[0] * (I[ofs] - memory[1]) / (sqrt(memory[2]) + eps);
}
extern "C"
void batch_norm(float * I, float * M, float * G, float * V, float * R, float eps, int ow, int oh, int oc){
    float *dev_I, *dev_M, *dev_G, *dev_V, *dev_R;
    // I: (ow * oh * oc), row major ordered
    // M, G, V, R: (oc)
    // R: (ow * oh * oc), row major ordered
    // todo: element-wise normalization
    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_I, ow * oh * oc * sizeof(float) );
    hipMalloc( (void**)&dev_M, oc * sizeof(float) );
    hipMalloc( (void**)&dev_G, oc * sizeof(float) );
    hipMalloc( (void**)&dev_V, oc * sizeof(float) );
    hipMalloc( (void**)&dev_R, ow * oh * oc * sizeof(float) );
    // copy the arrays to the GPU
    hipMemcpy( dev_I, I, ow * oh * oc * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_M, M, oc * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_G, G, oc * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_V, V, oc * sizeof(float), hipMemcpyHostToDevice );
    // block = channel, thread over pixels
    int BLOCKS_PER_CHANNEL = ceil(float(ow*oh)/float(THREADS_PER_BLOCK));
    bn<<<oc*BLOCKS_PER_CHANNEL,THREADS_PER_BLOCK>>>(dev_I, dev_M, dev_G, dev_V, dev_R, eps, ow, oh, oc);
    // copy the array back from the GPU to the CPU
    hipMemcpy( R, dev_R, ow * oh * oc * sizeof(float), hipMemcpyDeviceToHost );
    // cleanup
    hipFree(dev_I); hipFree(dev_M); hipFree(dev_G); hipFree(dev_V); hipFree(dev_R);
}





__global__ void mp(float *I, float *R, int iw, int ih, int kw, int kh, int sw, int sh, int ow, int oh, int oc){
    // input stationary
    int BLOCKS_PER_CHANNEL = ceil(float(ow * oh)/float(THREADS_PER_BLOCK));
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int pid = bid % BLOCKS_PER_CHANNEL; // pixel block index (within channel)
    int cid = bid / BLOCKS_PER_CHANNEL; // output channel index
    // compute block index in output pixel dimension
    int ofs = pid * THREADS_PER_BLOCK;
    // handle boundary
    if (tid >= ((ow * oh - ofs < THREADS_PER_BLOCK)? (ow * oh - ofs) : THREADS_PER_BLOCK)) return;
    // retrieve output pixel
    int w = (ofs + tid)/oh;
    int h = (ofs + tid)%oh;
    int w_ofs = w*sw;
    int h_ofs = h*sh;
    // apply pooling
    float v = -1e20;
    float input;
    int lw = (kw < iw-w_ofs)? kw : (iw-w_ofs);
    int lh = (kh < ih-h_ofs)? kh : (ih-h_ofs);
    for (int i=0; i<lw; i++){
        for (int j=0; j<lh; j++){
            int idx = INDEX_ROW_MAJOR_3(w_ofs+i,h_ofs+j,cid, iw,ih,oc);
            input = I[idx];
            v = ((input > v)? input : v);
        }
    }
    R[INDEX_ROW_MAJOR_3(w,h,cid, ow,oh,oc)] = v;
}
extern "C"
void max_pool(float * I, float * R, int iw, int ih, int kw, int kh, int sw, int sh, int ow, int oh, int oc) {
    float *dev_I, *dev_R;
    // I: (iw * ih * oc), row major ordered
    // R: (ow * oh * oc), row major ordered
    // todo: max-pooling
    // kernel function: pooling for a single sliding window
    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_I, iw * ih * oc * sizeof(float) );
    hipMalloc( (void**)&dev_R, ow * oh * oc * sizeof(float) );
    // copy the arrays to the GPU
    hipMemcpy( dev_I, I, iw * ih * oc * sizeof(float), hipMemcpyHostToDevice );
    // within a block, thread over output pixels
    int BLOCKS_PER_CHANNEL = ceil(float(ow * oh)/float(THREADS_PER_BLOCK));
    mp<<<oc*BLOCKS_PER_CHANNEL,THREADS_PER_BLOCK>>>(dev_I, dev_R, iw, ih, kw, kh, sw, sh, ow, oh, oc);
    // copy the array back from the GPU to the CPU
    hipMemcpy( R, dev_R, ow * oh * oc * sizeof(float), hipMemcpyDeviceToHost );
    // cleanup
    hipFree(dev_I); hipFree(dev_R);
}
